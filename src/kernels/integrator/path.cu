#include "hip/hip_runtime.h"
#include "path.cuh"

#include "common.cuh"

namespace kernel {

namespace {

CU_DEVICE glm::vec3 Trace(const PathTracer::Params &params, Ray ray, SamplerState &sampler) {
    AccelHitInfo hit_info;
    if (!params.scene.accel->Intersect(ray, hit_info)) {
        // TODO - envmap
        return glm::vec3(0.0f);
    }
    auto surface = params.scene.instances[hit_info.instance_id].GetShadingSurface(hit_info);

    glm::vec3 color = surface.bsdf.emission;
    glm::vec3 throughput(1.0f);
    for (uint32_t depth = 0; depth < params.max_depth; depth++) {
        Frame frame(surface.vertex.normal);
        auto wo = frame.ToLocal(-ray.direction);

        if (!surface.bsdf.IsDelta()) {
            float light_sample_pdf;
            auto light = params.scene.light_sampler.Sample(surface.vertex.position, sampler.Next1D(), light_sample_pdf);
            auto light_samp = light.Sample(surface.vertex.position, sampler.Next2D());
            light_samp.pdf *= light_sample_pdf;
            light_samp.weight /= light_sample_pdf;
            if (light_samp.pdf > 0.0f) {
                Ray shadow_ray(surface.vertex.position, light_samp.dir);
                shadow_ray.tmax = light_samp.dist - Ray::kShadowRayEps;
                if (!params.scene.accel->Occlude(shadow_ray)) {
                    auto wi = frame.ToLocal(light_samp.dir);
                    float mis_weight = 1.0f;
                    if (!light.IsDelta()) {
                        auto bsdf_pdf = surface.bsdf.Pdf(wo, wi);
                        mis_weight = PowerHeuristic(light_samp.pdf, bsdf_pdf);
                    }
                    color += throughput * mis_weight * light_samp.weight * surface.bsdf.Eval(wo, wi);
                }
            }
        }

        auto bsdf_samp = surface.bsdf.Sample(wo, sampler.Next1D(), sampler.Next2D());
        if (bsdf_samp.pdf == 0.0f) {
            break;
        }
        throughput *= bsdf_samp.weight;
        ray = Ray(surface.vertex.position, frame.ToWorld(bsdf_samp.wi));

        if (!params.scene.accel->Intersect(ray, hit_info)) {
            // TODO - envmap
            break;
        }

        surface = params.scene.instances[hit_info.instance_id].GetShadingSurface(hit_info);
        if (surface.bsdf.emission != glm::vec3(0.0f) && glm::dot(ray.direction, surface.vertex.normal) < 0.0f) {
            float mis_weight = 1.0f;
            if (bsdf_samp.lobe.type != BsdfLobe::Type::eSpecular) {
                const auto &light = params.scene.instances[hit_info.instance_id].light;
                auto light_pdf = surface.vertex.pdf
                    * PathToSolidAngleJacobian(ray.origin, surface.vertex.position, surface.vertex.normal)
                    * params.scene.light_sampler.Pdf(ray.origin, light);
                mis_weight = PowerHeuristic(bsdf_samp.pdf, light_pdf);
            }
            color += throughput * mis_weight * surface.bsdf.emission;
        }

        float rr_prop = glm::clamp(Luminance(throughput), 0.01f, 0.95f);
        if (sampler.Next1D() > rr_prop) {
            break;
        }
        throughput /= rr_prop;
    }

    return color;
}

CU_GLOBAL void RenderKernel(PathTracer::Params params) {
    glm::uvec2 pixel_coord(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    if (pixel_coord.x >= params.screen_width || pixel_coord.y >= params.screen_height) {
        return;
    }
    auto pixel_index = (params.screen_height - 1 - pixel_coord.y) * params.screen_width + pixel_coord.x;

    auto sampler = SamplerState::Create(pixel_index, params.spp);

    auto subpixel = params.spp == 1 ? glm::vec2(0.5f) : sampler.Next2D();
    auto ray = params.scene.camera.SampleRay(static_cast<float>(params.screen_width) / params.screen_height,
        (glm::vec2(pixel_coord) + subpixel) / glm::vec2(params.screen_width, params.screen_height), sampler.Next2D());

    auto color = Trace(params, ray, sampler);
    if (glm::any(glm::isnan(color)) || glm::any(glm::isinf(color))) {
        color = glm::vec3(0.0f);
    }
    auto prev_color = params.output[pixel_index];
    auto mixed_color = glm::mix(prev_color, glm::vec4(color, 1.0f), 1.0f / params.spp);
    params.output[pixel_index] = mixed_color;
}

}

void PathTracer::Render(const Params &params) {
    dim3 threads(16, 16, 1);
    dim3 grids((params.screen_width + threads.x - 1) / threads.x, (params.screen_height + threads.y - 1) / threads.y);
    RenderKernel<<<grids, threads>>>(params);
    auto r = hipDeviceSynchronize();
    assert(r == 0);
}

}
