#include "hip/hip_runtime.h"
#include "accel_build.cuh"

#include <memory>

#include <thrust/sort.h>

#include "cuda_helpers/buffer.hpp"

namespace kernel {

namespace {

constexpr float kMortonCodeResolution = 1024.0f;

constexpr uint32_t kThreads = 32;

CU_DEVICE uint32_t MortonCode3(uint32_t x) {
    x = (x ^ (x << 16)) & 0xff0000ff;
	x = (x ^ (x << 8)) & 0x0300f00f;
	x = (x ^ (x << 4)) & 0x030c30c3;
	x = (x ^ (x << 2)) & 0x09249249;
    return x;
}

CU_GLOBAL void CalcMortonCode(uint64_t *codes, const Bbox *bboxes, Bbox merged_bbox, uint32_t num_primitives) {
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_primitives) {
        return;
    }

    auto center = (bboxes[index].pmin + bboxes[index].pmax) * 0.5f;
    auto scale = merged_bbox.pmax - merged_bbox.pmin;
    auto p = (center - merged_bbox.pmin) / scale;

    auto x = MortonCode3(fmin(p.x * kMortonCodeResolution, kMortonCodeResolution - 1));
    auto y = MortonCode3(fmin(p.y * kMortonCodeResolution, kMortonCodeResolution - 1));
    auto z = MortonCode3(fmin(p.z * kMortonCodeResolution, kMortonCodeResolution - 1));
    uint64_t morton_code = (x << 2) | (y << 1) | z;

    codes[index] = (morton_code << 32) | index;
}

CU_GLOBAL void FillLeafNodes(AccelNode *nodes, const uint64_t *codes, uint32_t num_primitives) {
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_primitives) {
        return;
    }

    nodes[index].lc_or_id = codes[index];
    nodes[index].rc = ~0u;
}

CU_DEVICE uint32_t Lcp(uint64_t a, uint64_t b) {
    return __clzll(a ^ b);
}

CU_DEVICE glm::uvec2 FindNodeRange(uint32_t index, const uint64_t *codes, uint32_t num_primitives) {
    if (index == 0) {
        return glm::uvec2(0, num_primitives - 1);
    }

    auto code = codes[index];
    auto l_lcp = Lcp(code, codes[index - 1]);
    auto r_lcp = Lcp(code, codes[index + 1]);
    
    auto d = l_lcp > r_lcp ? -1 : 1;
    auto min_lcp = glm::min(l_lcp, r_lcp);
    uint32_t step = 1;
    uint32_t j_lcp;
    do {
        step <<= 1;
        int j = index + d * step;
        j_lcp = 0;
        if (j >= 0 && j < num_primitives) {
            j_lcp = Lcp(code, codes[j]);
        }
    } while (j_lcp > min_lcp);

    auto l = step >> 1;
    auto r = glm::min(step, d == -1 ? index : num_primitives - 1 - index);
    while (l < r) {
        auto mid = l + ((r - l) >> 1) + 1;
        auto j = index + d * mid;
        auto j_lcp = Lcp(code, codes[j]);
        if (j_lcp < min_lcp) {
            r = mid - 1;
        } else {
            l = mid;
        }
    }

    r = index + d * l;
    l = index;
    return l <= r ? glm::uvec2(l, r) : glm::uvec2(r, l);
}

CU_DEVICE uint32_t FindNodeLeftChild(uint32_t index, glm::uvec2 range, const uint64_t *codes, uint32_t num_primitives) {
    auto l_code = codes[range.x];
    auto r_code = codes[range.y];
    auto node_lcp = Lcp(l_code, r_code);

    auto l = range.x;
    auto r = range.y;
    while (l < r) {
        auto mid = l + ((r - l) >> 1);
        auto m_lcp = Lcp(l_code, codes[mid]);
        if (m_lcp > node_lcp) {
            l = mid + 1;
        } else {
            r = mid;
        }
    }

    return l - 1;
}

CU_GLOBAL void BuildInternalNodes(AccelNode *nodes, uint32_t *parents, const uint64_t *codes, uint32_t num_primitives) {
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_primitives - 1) {
        return;
    }

    auto range = FindNodeRange(index, codes, num_primitives);
    auto lc = FindNodeLeftChild(index, range, codes, num_primitives);
    auto rc = lc + 1;

    if (lc == range.x) {
        lc += num_primitives - 1;
    }
    if (rc == range.y) {
        rc += num_primitives - 1;
    }

    nodes[index].lc_or_id = lc;
    nodes[index].rc = rc;
    parents[lc] = index;
    parents[rc] = index;
}

CU_GLOBAL void InitInternalNodesBbox(Bbox *bboxes, Bbox merged_bbox, uint32_t num_primitives) {
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_primitives - 1) {
        return;
    }

    bboxes[index].pmin = merged_bbox.pmax;
    bboxes[index].pmax = merged_bbox.pmin;
}

CU_DEVICE float atomicMinFloat(float *addr, float value) {
    return !signbit(value) ? __int_as_float(atomicMin(reinterpret_cast<int *>(addr), __float_as_int(value))) :
        __uint_as_float(atomicMax(reinterpret_cast<uint32_t *>(addr), __float_as_uint(value)));
}

CU_DEVICE float atomicMaxFloat(float *addr, float value) {
    return !signbit(value) ? __int_as_float(atomicMax(reinterpret_cast<int *>(addr), __float_as_int(value))) :
        __uint_as_float(atomicMin(reinterpret_cast<uint32_t *>(addr), __float_as_uint(value)));
}

CU_GLOBAL void CalcInternalNodesBbox(const uint32_t *parents, Bbox *bboxes, uint32_t num_primitives) {
    uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= num_primitives) {
        return;
    }

    uint32_t u = num_primitives - 1 + index;
    uint32_t pa = parents[u];
    auto bbox = bboxes[u];
    while (u != 0) {
        atomicMinFloat(&bboxes[pa].pmin.x, bbox.pmin.x);
        atomicMinFloat(&bboxes[pa].pmin.y, bbox.pmin.y);
        atomicMinFloat(&bboxes[pa].pmin.z, bbox.pmin.z);
        atomicMaxFloat(&bboxes[pa].pmax.x, bbox.pmax.x);
        atomicMaxFloat(&bboxes[pa].pmax.y, bbox.pmax.y);
        atomicMaxFloat(&bboxes[pa].pmax.z, bbox.pmax.z);

        u = pa;
        pa = parents[u];
    }
}

}

void BuildAccel(AccelNode *nodes, Bbox *bboxes, Bbox merged_bbox, uint32_t num_primitives) {
    auto num_internal_nodes = num_primitives - 1;

    auto morton_codes_buffer = std::make_unique<CuBuffer>(sizeof(uint64_t) * num_primitives);
    auto morton_codes = morton_codes_buffer->TypedGpuData<uint64_t>();
    auto bboxes_leaf = bboxes + num_internal_nodes;
    CalcMortonCode<<<(num_primitives + kThreads - 1) / kThreads, kThreads>>>(
        morton_codes, bboxes_leaf, merged_bbox, num_primitives);

    thrust::sort_by_key(thrust::device, morton_codes, morton_codes + num_primitives, bboxes_leaf);

    auto nodes_leaf = nodes + num_internal_nodes;
    FillLeafNodes<<<(num_primitives + kThreads - 1) / kThreads, kThreads>>>(nodes_leaf, morton_codes, num_primitives);

    auto parents_buffer = std::make_unique<CuBuffer>(sizeof(uint32_t) * (num_primitives + num_internal_nodes));
    auto parents = parents_buffer->TypedGpuData<uint32_t>();
    BuildInternalNodes<<<(num_internal_nodes + kThreads - 1) / kThreads, kThreads>>>(
        nodes, parents, morton_codes, num_primitives);

    InitInternalNodesBbox<<<(num_internal_nodes + kThreads - 1) / kThreads, kThreads>>>(
        bboxes, merged_bbox, num_primitives);
    CalcInternalNodesBbox<<<(num_primitives + kThreads - 1) / kThreads, kThreads>>>(parents, bboxes, num_primitives);
}

}
